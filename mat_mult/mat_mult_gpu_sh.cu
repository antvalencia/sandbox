
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>

#define TILES 2


__global__ void
mat_mult(float *m1d, float *m2d, float *mpd, const int DIM) {
	__shared__ float m1s [TILES][TILES];
	__shared__ float m2s [TILES][TILES];

	unsigned int col = (TILES * blockIdx.x) + threadIdx.x;
	unsigned int row = (TILES * blockIdx.y) + threadIdx.y;

	for (int m=0; m<(DIM/TILES); m++) {
		m1s[threadIdx.y][threadIdx.x] =  m1d[(row * DIM) + ((m * TILES) + threadIdx.x)];
		m2s[threadIdx.y][threadIdx.x] =  m2d[(((m * TILES) + threadIdx.y) * DIM) + col];
		__syncthreads();

		for (int k=0; k<TILES; k++)
			mpd[(row * DIM) + col] += (m1s[threadIdx.x][k] * m2s[k][threadIdx.y]);
		__syncthreads();
	}
}

int main () {
	const int DIM = 512;
	const bool PRINT_RESULT = false;
	float m1_h[DIM][DIM], m2_h[DIM][DIM], m_prod_h[DIM][DIM];
	float *m1_d, *m2_d, *m_prod_d;
	int i, j;
	srand(time(NULL));

	// initialize matrices on host
	for (i=0; i<DIM; i++) {
		for (j=0; j<DIM; j++) {
			m1_h[i][j] = rand() % 10;
			m2_h[i][j] = rand() % 10;
		}
	}

	// allocate memory for device matrices
	hipMalloc((void **) &m1_d, DIM * DIM * sizeof(int));
	hipMalloc((void **) &m2_d, DIM * DIM * sizeof(int));

	// copy host matrix to device
	hipMemcpy(m1_d, m1_h, DIM * DIM * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(m2_d, m2_h, DIM * DIM * sizeof(int), hipMemcpyHostToDevice);

	// allocate memory for product matrix
	hipMalloc((void **) &m_prod_d, DIM * DIM * sizeof(int));

	// call kernel
	dim3 dimGrid(DIM/TILES, DIM/TILES, 1);
	dim3 dimBlock(TILES, TILES, 1);

        float exec_t;
        hipEvent_t s_t, e_t;
        hipEventCreate(&s_t);
        hipEventCreate(&e_t);
        hipEventRecord(s_t, 0);
	mat_mult <<<dimGrid,dimBlock>>> (m1_d, m2_d, m_prod_d, DIM);

	hipMemcpy(m_prod_h, m_prod_d, DIM * DIM * sizeof(int), hipMemcpyDeviceToHost);

        hipEventRecord(e_t, 0);
        hipEventSynchronize(e_t);
        hipEventElapsedTime(&exec_t, s_t, e_t);

        if (PRINT_RESULT) {
		for (i=0; i<DIM; i++) {
			for (j=0; j<DIM; j++) {
				printf("%f   ", m_prod_h[i][j]);
			}
			printf("\n");
		}
	}
	printf("GPU-(shr)-dim: %iX%i; exec time: %3.5fms \n", DIM, DIM, exec_t);
}
